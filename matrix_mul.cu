
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

//for sequential
// void mulMat(int mat1[][C1], int mat2[][C2])
// {
//     int rslt[R1][C2];

//     cout << "Multiplication of given two matrices is:\n";

//     for (int i = 0; i < R1; i++) {
//         for (int j = 0; j < C2; j++) {
//             rslt[i][j] = 0;

//             for (int k = 0; k < R2; k++) {
//                 rslt[i][j] += mat1[i][k] * mat2[k][j];
//             }

//             cout << rslt[i][j] << "\t";
//         }

//         cout << endl;
//     }
// }

__global__ void CudaMultiplication(int* x, int* y, int* z, int N1, int N2, int M1, int M2){
      int row = blockIdx.x * blockDim.x + threadIdx.x;
      int col = blockIdx.y * blockDim.y + threadIdx.y;
      int val = 0;


      for(int k=0; k<N2; k++){
          val += x[row*N2+k]*y[k*M2+col] ;
      }

      z[row*M2+col]=val;
}

int main(){

    int N1 = 3;
    int N2 = 4;
    int M1 = 4;
    int M2 = 1;

    int *x = (int *)malloc(sizeof(int)*N1*N2);
    int *y = (int *)malloc(sizeof(int)*M1*M2);
    int *z = (int *)malloc(sizeof(int)*N1*M2);

    for(int i=0;i<N1;i++){
      for(int j=0;j<N2;j++) {
          x[i*N2+j]=1;
      }
    }

    for(int i=0;i<M1;i++){
      for(int j=0;j<M2;j++) {
          y[i*M2+j]=1;
      }
    }

    int *a , *b , *c;
    hipMalloc(&a , sizeof(int)*N1*N2);
    hipMalloc(&b , sizeof(int)*M1*M2);
    hipMalloc(&c , sizeof(int)*N1*M2);

    hipMemcpy(a, x, sizeof(int)*N1*N2, hipMemcpyHostToDevice);
    hipMemcpy(b, y, sizeof(int)*M1*M2, hipMemcpyHostToDevice);

    dim3 th(N1,M2);
    CudaMultiplication<<<1, th>>>(a, b, c, N1, N2, M1, M2);

    hipMemcpy(z, c, sizeof(int)*N1*M2, hipMemcpyDeviceToHost);

    for(int i=0;i<N1;i++){
        for(int j=0;j<M2;j++){
            cout<<z[i*M2+j]<<" ";
        }
        cout<<endl;
    }
    return 0;
}